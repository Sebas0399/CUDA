#include <iostream>
#include <hip/hip_runtime.h>
#define VECTOR_ELEMENTS 1024
__global__
void vecAdd(float *d_A,float  *d_B,float *d_C,int n){
    int index=threadIdx.x+blockDim.x+blockIdx.x;
    if(index<n){
        d_C[index]=d_A[index]+d_B[index];
    }
}
int main() {
    //HOST
    float *h_A=new float [VECTOR_ELEMENTS];
    float *h_B=new float [VECTOR_ELEMENTS];

    float *h_C=new float [VECTOR_ELEMENTS];
//DEVICE
    float *d_A,*d_B,*d_C,*xx;
    int size=VECTOR_ELEMENTS*sizeof (float );
    hipMalloc(&d_A,size);
    hipMalloc(&d_B,size);
    hipMalloc(&d_C,size);
    hipMalloc(&xx,1024*1014*1024);

//Inicializar vectores en el host
    for (int i=0;i<VECTOR_ELEMENTS;i++){
        h_A[i]=1.f;
        h_B[i]=2.f;
        h_C[i]=0.0f;

    }
    //copiar host to device
    hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
    //executar kernel
    vecAdd<<<4,256>>>(d_A,d_B,d_C,VECTOR_ELEMENTS);
    //copiar device to host
    hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
    for (int i=0;i<10;i++){
        printf("%.0f, ",h_C[i]);
    }
    return 0;
}
